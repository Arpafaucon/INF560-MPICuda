#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// includes
#include <helper_functions.h> // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>	  // helper functions for CUDA error checking and initialization

extern "C"
{
#include <hip/hip_runtime.h>
}
#define MEMSIZE 30

__global__ void kern_compute_string(char *res, char *a, char *b, char *c, int length)
{
	int i;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < length)
	{
		res[i] = a[i] + b[i] + c[i];
	}
}

/* Function computing the final string to print */
void compute_string(char *res, char *a, char *b, char *c, int length)
{
	int i;

	for (i = 0; i < length; i++)
	{
		res[i] = a[i] + b[i] + c[i];
	}
}

extern "C" int nameOfFunction()
{

	char *res;

	char a[30] = {40, 70, 70, 70, 80, 0, 50, 80, 80, 70, 70, 0, 40, 80, 79,
				  70, 0, 40, 50, 50, 0, 70, 80, 0, 30, 50, 30, 30, 0, 0};
	char b[30] = {10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10,
				  10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 0, 0};
	char c[30] = {22, 21, 28, 28, 21, 22, 27, 21, 24, 28, 20, 22, 20, 24, 22,
				  29, 22, 21, 20, 25, 22, 25, 20, 22, 27, 25, 28, 25, 0, 0};

	res = (char *)malloc(30 * sizeof(char));

	/* This function call should be programmed in CUDA */
	/* -> need to allocate and transfer data to/from the device */
	char *d_a, *d_b, *d_c, *d_res;
	dim3 gridDim;
	gridDim.x = 8;
	dim3 blockDim;
	blockDim.x = 8;

	checkCudaErrors(hipMalloc((void **)&d_a, MEMSIZE * sizeof(char)));
	checkCudaErrors(hipMalloc((void **)&d_b, MEMSIZE * sizeof(char)));
	checkCudaErrors(hipMalloc((void **)&d_c, MEMSIZE * sizeof(char)));
	checkCudaErrors(hipMalloc((void **)&d_res, MEMSIZE * sizeof(char)));

	//initialize the device memory
	checkCudaErrors(hipMemcpy(d_a, a, MEMSIZE,	hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_b, b, MEMSIZE,	hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_c, c, MEMSIZE,	hipMemcpyHostToDevice));
	
	kern_compute_string<<<gridDim, blockDim>>>(d_res, d_a, d_b, d_c, MEMSIZE);
	
	checkCudaErrors(hipMemcpy(res, d_res, MEMSIZE,	hipMemcpyDeviceToHost));
	
	// compute_string(res, a, b, c, MEMSIZE);

	printf("%s\n", res);

	return 0;
}
